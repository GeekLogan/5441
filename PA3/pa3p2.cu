// Template for Programming Assignment 2
// Use "module load cuda" to enable compilation with the Nvidia C compiler nvcc
// Use "nvcc -O3" to compile code; this can be done even on OSC login node (does not have a GPU)
// To execute compiled code, you must either use a batch submission to run on a node with GPU
// or obtain an interactive GPU-node by using: qsub -I -l walltime=0:59:00 -l nodes=1:gpus=1


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <sys/time.h>
#define threshold 1e-8
// #define n (4096)
// Change n to 4096 for final testing; 
#define n (4096)
#define BLOCK_SIZE 16
// n is set to 256 since execution time of single thread template version is excessive
void init(void);
void ref(void);
void compare(int N, double *wref, double *w);
__global__ void test_kernel(int N, double *A, double *B, double *C);
double rtclock(void);

double a[n][n],b[n][n],c[n][n],cref[n][n];

int main(){

double clkbegin, clkend, t;
double *Ad,*Bd,*Cd;
int size;

  printf("Matrix Size = %d\n",n);

  init();
  clkbegin = rtclock();
  ref();
  clkend = rtclock();
  t = clkend-clkbegin;
  printf("Seq: Approx GFLOPS: %.1f ; Time = %.3f sec; cref[n/2][n/2-1] = %f; \n",
2.0*n*n*n/t/1e9,t,cref[n/2][n/2-1]);

  
  size = sizeof(double)*n*n;
  hipMalloc((void **) &Ad,size);
  hipMalloc((void **) &Bd,size);
  hipMalloc((void **) &Cd,size);
  hipMemcpy(Ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(Bd,b,size,hipMemcpyHostToDevice);
  clkbegin = rtclock();
  
  dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
  dim3 grid( (n + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y );
  test_kernel<<<grid, threads>>>(n,Ad,Bd,Cd);
  if (hipDeviceSynchronize() != hipSuccess) 
    printf ("Error return for test_kernel: Was execution done on a node with a GPU?\n");
  else
  {
   clkend = rtclock();
   t = clkend-clkbegin;
   hipMemcpy(c,Cd,size,hipMemcpyDeviceToHost);
   hipFree(Ad); hipFree(Bd); hipFree(Cd);
    printf("GPU: Approx GFLOPS: %.1f ; Time = %.3f sec; c[n/2][n/2-1] = %f; \n",
     2.0*n*n*n/t/1e9,t,c[n/2][n/2-1]);
    printf("Correctness Check for GPU solution:\n");
    compare(n, (double *) c,(double *) cref);
  }
}

__global__ void test_kernel(int N, double *A, double *B, double *C)
{
// Template version uses only one thread, which does all the work
// This must be changed (and the launch parameters) to exploit GPU parallelism
// You can make any changes; only requirement is that correctness test passes

int k, kt;

const int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;
const int j = blockIdx.y * BLOCK_SIZE + threadIdx.y;
const int col_offset = blockIdx.y*BLOCK_SIZE+threadIdx.x;

__shared__ double a_buffer[ BLOCK_SIZE ][ BLOCK_SIZE ];
__shared__ double b_buffer[ BLOCK_SIZE ][ BLOCK_SIZE ];

double sum = 0;

for(kt=0;kt<N;kt+=BLOCK_SIZE){

	a_buffer[ threadIdx.y ][ threadIdx.x ] = A[ (kt+threadIdx.y)*N+i ];
	b_buffer[ threadIdx.y ][ threadIdx.x ] = B[ (kt+threadIdx.y)*N+col_offset ];

	__syncthreads();

	for(k=0;k<BLOCK_SIZE;k++)
		sum += a_buffer[k][threadIdx.x] * b_buffer[k][threadIdx.y];

	__syncthreads();

}

C[j*N+i] = sum;

}

void ref(void)
{
int i,j,k;

  for (j=0;j<n;j++)
   for (k=0;k<n;k++)
    for (i=0;i<n;i++)
      cref[j][i] += a[k][i]*b[k][j];
}

void init(void)
{
int i,j;
for(i=0;i<n;i++)
 for(j=0;j<n;j++) 
 { c[i][j] = 0.0; 
   cref[i][j] = 0.0; 
   a[i][j] = drand48();
   b[i][j] = drand48();
 }
}

void compare(int N, double *wref, double *w)
{
double maxdiff,this_diff;
int numdiffs;
int i,j;
  numdiffs = 0;
  maxdiff = 0;
  for (i=0;i<N;i++)
   for (j=0;j<N;j++)
    {
     this_diff = wref[i*N+j]-w[i*N+j];
     if (this_diff < 0) this_diff = -1.0*this_diff;
     if (this_diff>threshold)
      { numdiffs++;
        if (this_diff > maxdiff) maxdiff=this_diff;
      }
    }
   if (numdiffs > 0)
      printf("%d Diffs found over threshold %f; Max Diff = %f\n",
               numdiffs,threshold,maxdiff);
   else
      printf("No differences found between reference and test versions\n");
}

double rtclock(void)
{
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday (&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d",stat);
  return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}
